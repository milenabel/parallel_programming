
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define threshold 0.0001

void checkCUDAError(const char *msg);

const int DSIZE = 1024;
hipEvent_t start, stop;
float tstart, elapsedTime;

// Matrix multiply kernel: C = A^T * B^T with 4-way unrolling along i
__global__ void mmtt_i4(const float *A, const float *B, float *C, int ds) {
    int row_base = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_base < ds) {
        for (int col = 0; col < ds; col++) {
            float value0 = 0.0f, value1 = 0.0f, value2 = 0.0f, value3 = 0.0f;

            for (int k = 0; k < ds; k++) {
                value0 += A[k * ds + (row_base + 0)] * B[col * ds + k];
                if (row_base + 1 < ds) value1 += A[k * ds + (row_base + 1)] * B[col * ds + k];
                if (row_base + 2 < ds) value2 += A[k * ds + (row_base + 2)] * B[col * ds + k];
                if (row_base + 3 < ds) value3 += A[k * ds + (row_base + 3)] * B[col * ds + k];
            }

            C[(row_base + 0) * ds + col] = value0;
            if (row_base + 1 < ds) C[(row_base + 1) * ds + col] = value1;
            if (row_base + 2 < ds) C[(row_base + 2) * ds + col] = value2;
            if (row_base + 3 < ds) C[(row_base + 3) * ds + col] = value3;
        }
    }
}

int main() {
    float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
    int i, j, k;

    h_A = new float[DSIZE * DSIZE];
    h_B = new float[DSIZE * DSIZE];
    h_C = new float[DSIZE * DSIZE];
    h_Cref = new float[DSIZE * DSIZE];

    for (i = 0; i < DSIZE * DSIZE; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
        h_C[i] = 0;
        h_Cref[i] = 0;
    }

    for (i = 0; i < DSIZE; i++) {
        for (k = 0; k < DSIZE; k++) {
            for (j = 0; j < DSIZE; j++) {
            //  h_Cref[i][j] += h_A[k][i]*h_B[j][k];
                h_Cref[i * DSIZE + j] += h_A[k * DSIZE + i] * h_B[j * DSIZE + k];
            }
        }
    }

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * DSIZE * sizeof(float));
    checkCUDAError("hipMalloc failure");
    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy H2D transfer failure");

    dim3 block(256, 1);
    dim3 grid((DSIZE + block.x - 1) / block.x, 1);

    printf("Matrix size: %d\n", DSIZE);

    for (int trial = 0; trial < 3; trial++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        // Launch kernel
        mmtt_i4<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
        checkCUDAError("GPU kernel launch failure");
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipDeviceSynchronize();
        // Copy results back to host
        hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(float), hipMemcpyDeviceToHost);
        checkCUDAError("hipMemcpy D2H");

        for (int i = 0; i < DSIZE * DSIZE; i++) {
            if (fabs((h_C[i] - h_Cref[i]) / h_Cref[i]) > threshold) {
                printf("Mismatch at %d: GPU %f CPU %f\n", i, h_C[i], h_Cref[i]);
                break;
            }
        }

        printf("Trial %d: Elapsed Time: %f ms, GFLOPS: %.2f\n", trial + 1, elapsedTime, 2.0e-6 * DSIZE * DSIZE * DSIZE / elapsedTime);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_Cref;

    return 0;
}

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
