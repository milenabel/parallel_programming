
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define threshold 0.0001

void checkCUDAError(const char *msg);

const int DSIZE = 1024;
hipEvent_t start, stop;
float tstart, elapsedTime;

// Matrix multiply kernel: C = A^T * B^T with 4-way unrolling along j
__global__ void mmtt_j4(const float *A, const float *B, float *C, int ds) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ds) {
        for (int col_base = 0; col_base < ds; col_base += 4) {
            float value0 = 0.0f, value1 = 0.0f, value2 = 0.0f, value3 = 0.0f;

            for (int k = 0; k < ds; k++) {
                float a_val = A[k * ds + row];
                value0 += a_val * B[(col_base + 0) * ds + k];
                value1 += a_val * B[(col_base + 1) * ds + k];
                value2 += a_val * B[(col_base + 2) * ds + k];
                value3 += a_val * B[(col_base + 3) * ds + k];
            }

            if (col_base + 0 < ds) C[row * ds + (col_base + 0)] = value0;
            if (col_base + 1 < ds) C[row * ds + (col_base + 1)] = value1;
            if (col_base + 2 < ds) C[row * ds + (col_base + 2)] = value2;
            if (col_base + 3 < ds) C[row * ds + (col_base + 3)] = value3;
        }
    }
}

int main() {
    float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
    int i, j, k;

    h_A = new float[DSIZE * DSIZE];
    h_B = new float[DSIZE * DSIZE];
    h_C = new float[DSIZE * DSIZE];
    h_Cref = new float[DSIZE * DSIZE];

    for (i = 0; i < DSIZE * DSIZE; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
        h_C[i] = 0;
        h_Cref[i] = 0;
    }

    for (i = 0; i < DSIZE; i++) {
        for (k = 0; k < DSIZE; k++) {
            for (j = 0; j < DSIZE; j++) {
            //  h_Cref[i][j] += h_A[k][i]*h_B[j][k];
                h_Cref[i * DSIZE + j] += h_A[k * DSIZE + i] * h_B[j * DSIZE + k];
            }
        }
    }

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * DSIZE * sizeof(float));
    checkCUDAError("hipMalloc failure");
    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy H2D transfer failure");

    dim3 block(256, 1);
    dim3 grid((DSIZE + block.x - 1) / block.x, 1);

    printf("Matrix size: %d\n", DSIZE);

    for (int trial = 0; trial < 3; trial++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        // Launch kernel
        mmtt_j4<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
        checkCUDAError("GPU kernel launch failure");
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipDeviceSynchronize();
        // Copy results back to host
        hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(float), hipMemcpyDeviceToHost);
        checkCUDAError("hipMemcpy D2H");

        for (int i = 0; i < DSIZE * DSIZE; i++) {
            if (fabs((h_C[i] - h_Cref[i]) / h_Cref[i]) > threshold) {
                printf("Mismatch at %d: GPU %f CPU %f\n", i, h_C[i], h_Cref[i]);
                break;
            }
        }

        printf("Trial %d: Elapsed Time: %f ms, GFLOPS: %.2f\n", trial + 1, elapsedTime, 2.0e-6 * DSIZE * DSIZE * DSIZE / elapsedTime);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_Cref;

    return 0;
}

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
